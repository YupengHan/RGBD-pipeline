
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#include <math.h>
#include <ctime>



__global__ void findwindow (bool* mask_img,
                            int* scores) {
    int wh_p = blockIdx.x * blockDim.x + threadIdx.x; // threadIdx.x;
    int ui   = blockIdx.y * blockDim.y + threadIdx.y; // 60
    int vi   = blockIdx.z * blockDim.z + threadIdx.z; // 80

    int wh = 16;
    int cwhp = wh_p;
    while(cwhp > 0) {
        wh*=2;
        cwhp--;
    }
    

    if (ui == 0 && vi == 0) {
        //printf("wh:　%d;   wh_p:　%d \n", wh, wh_p);
    }


    int score_id = wh_p * 60 * 80 + ui * 80 + vi;

    int start_u = ui*8; //60 480
    int start_v = vi*8; //80 640
    int sc = 0;
    if (start_u + wh >= 480 || start_v + wh >= 640) {
        scores[score_id] = 0;
        return;
    }

    for (int cu = ui*8; cu < ui*8+wh; cu++) {
        for (int cv = vi*8; cv < vi*8+wh; cv++) {

            int mask_id = cu * 640 + cv;

            if (mask_img[mask_id]) {
                sc = sc+1;   
                //printf("cu: %d;   cv: %d;   mask_id: %d\n" , cu, cv, mask_id);
            }
        }
    }
    scores[score_id] = sc;

    if (sc > 0 && ui == 0 && vi == 0) {
        //printf("sc: %d            wh:　%d ui:　%d vi:　%d \n", sc, wh, ui, vi);
    }

}


__global__ void loop_2d_bbox (int* scores,
                            int* uvl) {
    //float mul = 1.3*1.3*1.3*1.3*1.3;
    float mul = 3*3*3*3*3;
    float final_c_best = 0.;
    //printf("start! \n");
    for (int idx = 0; idx < 4; idx++) {
        //mul /= 3.5;
        mul /= 3;
        int icbest = 0;
        int csu = 0, csv = 0, cwh = 0;
        for (int i = 0; i < 60; i++) {
            for (int j = 0; j < 80; j++) {
                if (scores[idx*4800+i*80+j] > icbest) {
                    icbest = scores[idx*4800+i*80+j];
                    csu = i*8;
                    csv = j*8;
                    cwh = idx;
                    //printf("icbest: %d  su: %d, sv: %d, wh: %d\n", icbest, csu, csv, cwh);
                }
            }
        }


        if (float(icbest)*mul > final_c_best) {
            final_c_best = float(icbest)*mul;
            uvl[0] = csu;
            uvl[1] = csv;
            uvl[2] = cwh;
            //printf("su: %d, sv: %d, wh: %d\n", csu, csv, cwh);
        }
    }
    
}

void rgb_window(bool* mask_img, int* scores, int* uvl) {
    //dim3 grid(4, 1, 1);
    //dim3 block(1, 60, 80);
    dim3 grid(4, 60, 80);
    dim3 block(1, 1, 1);
    
    findwindow<<<grid, block>>>(mask_img, scores);
    
    
    
    dim3 loop_grid(1, 1, 1);
    dim3 loop_block(1, 1, 1);
    loop_2d_bbox<<<loop_grid, loop_block>>>(scores, uvl);
    
    
    
    hipDeviceSynchronize();

}







__global__ void para_find_loc (float* pts,
                            int ptnum,
                            int* scores,
                            float* xyz_limits) {
    int d_ix = blockIdx.x * blockDim.x + threadIdx.x;
    int d_iy = blockIdx.y * blockDim.y + threadIdx.y;
    int d_iz = blockIdx.z * blockDim.z + threadIdx.z;
    //printf("d_ix: %d d_iy: %d d_iz: %d\n", d_ix, d_iy, d_iz);
    float start_x = xyz_limits[0];
    float start_y = xyz_limits[2];
    float start_z = xyz_limits[4];


    //printf("start_x: %.0f start_y: %.0f start_z: %.0f\n", start_x, start_y, start_z);

    float end_x = xyz_limits[1];
    float end_y = xyz_limits[3];
    float end_z = xyz_limits[5];

    
    float cx = start_x + d_ix*10;
    float cy = start_y + d_iy*10;
    float cz = start_z + d_iz*10;

    if (cx > end_x || cy > end_y || cz > end_z) {
        //printf("cx: %.0f cy: %.0f cz: %.0f end_x: %.0f end_y: %.0f end_z: %.0f ", cx, cy, cz, end_x, end_y, end_z);
        scores[d_ix*100*400+d_iy*400+d_iz] = 0;
        return;
    }
    //printf("cx: %.0f cy: %.0f cz: %.0f end_x: %.0f end_y: %.0f end_z: %.0f \n", cx, cy, cz, end_x, end_y, end_z);
    
    
    
    int cnt = 0;
    for(int i = 0; i < ptnum; i++) {
        float tx = pts[i*3];
        float ty = pts[i*3+1];
        float tz = pts[i*3+2];
        if (tz > cz) continue;
        float d2c = sqrt((tx-cx)*(tx-cx) + (ty-cy)*(ty-cy) + (tz-cz)*(tz-cz));
        //printf("tx: %.0f ty: %.0f tz: %.0f          d2c: %.0f\n", tx, ty, tz, d2c);  
        
        /*
        if (d2c < 1000) {
            printf("tx: %.0f ty: %.0f tz: %.0f          d2c: %.0f\n", tx, ty, tz, d2c);  
        }
        */
        //printf("tx: %.0f ty: %.0f tz: %.0f cx: %.0f cy: %.0f cz: %.0f \n", tx, ty, tz, cx, cy, cz);
        //printf("tx: %.0f ty: %.0f tz: %.0f          d2c: %.0f\n", tx, ty, tz, d2c);
        
        
        
        if (d2c >= 50 && d2c <= 53 ) {
            cnt += 1;
        }
    }
    scores[d_ix*100*400+d_iy*400+d_iz] = cnt;
}




__global__ void find_best_score (int* scores,
                                float* xyz_limits,
                                float* device_pred_xyz) {
    int c_best = 0;
    device_pred_xyz[0] = -10000;
    device_pred_xyz[1] = -10000;
    device_pred_xyz[2] = -10000;


    int ixmax = int((xyz_limits[1] - xyz_limits[0])/10);
    if (ixmax > 100) ixmax = 100;
    int iymax = int((xyz_limits[3] - xyz_limits[2])/10);
    if (iymax > 100) iymax = 100;
    int izmax = int((xyz_limits[5] - xyz_limits[4])/10);
    //if (izmax > 400) izmax = 400;
    if (izmax > 100) izmax = 100;
    printf("ixmax : %d;  iymax : %d;  izmax : %d\n", ixmax, iymax, izmax);

    for (int ix = 0; ix < ixmax; ix++) {
        for (int iy = 0; iy < iymax; iy++) {
            for (int iz = 0; iz < izmax; iz++) {
                //c_best = c_best > scores[ix*100*400+iy*400+iz] ? c_best : scores[ix*100*400+iy*400+iz];
                if (c_best < scores[ix*100*400+iy*400+iz]) {
                    c_best = scores[ix*100*400+iy*400+iz];
                    device_pred_xyz[0] = xyz_limits[0] + 10*ix;
                    device_pred_xyz[1] = xyz_limits[2] + 10*iy;
                    device_pred_xyz[2] = xyz_limits[4] + 10*iz;
                    //printf("Score: %d    x: %.0f    y: %.0f      z:%.0f \n", c_best, device_pred_xyz[0], device_pred_xyz[1], device_pred_xyz[2]);
                }
                
            }
        }
    }
}

void find_loc(float* pts, int ptnum, int* scores, float* xyz_limits, float* device_pred_xyz) {

    
    //dim3 grid(10, 100, 1);
    //dim3 block(10, 1, 400);
    
    dim3 grid(100, 100, 2);
    dim3 block(1, 1, 50);
    std::clock_t start, end;
    start = std::clock();
    para_find_loc<<<grid, block>>>(pts, ptnum, scores, xyz_limits);
    end = std::clock();
    printf("para_find_loc: %.3f ms\n", 1000. * (end - start)/CLOCKS_PER_SEC);
    
    start = std::clock();
    find_best_score<<<1, 1>>>(scores, xyz_limits, device_pred_xyz);
    end = std::clock();
    printf("find_best_score: %.3f ms\n", 1000. * (end - start)/CLOCKS_PER_SEC);




    

    hipDeviceSynchronize();
}





__global__ void para_find_loc_fine (float* pts,
                            int ptnum,
                            int* scores,
                            float* xyz_limits) {
    int d_ix = blockIdx.x * blockDim.x + threadIdx.x;
    int d_iy = blockIdx.y * blockDim.y + threadIdx.y;
    int d_iz = blockIdx.z * blockDim.z + threadIdx.z;
    //printf("d_ix: %d d_iy: %d d_iz: %d\n", d_ix, d_iy, d_iz);
    float start_x = xyz_limits[0];
    float start_y = xyz_limits[2];
    float start_z = xyz_limits[4];


    //printf("start_x: %.0f start_y: %.0f start_z: %.0f\n", start_x, start_y, start_z);

    float end_x = xyz_limits[1];
    float end_y = xyz_limits[3];
    float end_z = xyz_limits[5];

    
    /*
    float cx = start_x + d_ix*10;
    float cy = start_y + d_iy*10;
    float cz = start_z + d_iz*10;
    */

    float cx = start_x + d_ix*5;
    float cy = start_y + d_iy*5;
    float cz = start_z + d_iz*5;

    if (cx > end_x || cy > end_y || cz > end_z) {
        //printf("cx: %.0f cy: %.0f cz: %.0f end_x: %.0f end_y: %.0f end_z: %.0f ", cx, cy, cz, end_x, end_y, end_z);
        scores[d_ix*100*400+d_iy*400+d_iz] = 0;
        return;
    }
    //printf("cx: %.0f cy: %.0f cz: %.0f end_x: %.0f end_y: %.0f end_z: %.0f \n", cx, cy, cz, end_x, end_y, end_z);
    
    
    
    int cnt = 0;
    for(int i = 0; i < ptnum; i++) {
        float tx = pts[i*3];
        float ty = pts[i*3+1];
        float tz = pts[i*3+2];
        if (tz > cz) continue;
        float d2c = sqrt((tx-cx)*(tx-cx) + (ty-cy)*(ty-cy) + (tz-cz)*(tz-cz));
        //printf("tx: %.0f ty: %.0f tz: %.0f          d2c: %.0f\n", tx, ty, tz, d2c);  
        
        /*
        if (d2c < 1000) {
            printf("tx: %.0f ty: %.0f tz: %.0f          d2c: %.0f\n", tx, ty, tz, d2c);  
        }
        */
        //printf("tx: %.0f ty: %.0f tz: %.0f cx: %.0f cy: %.0f cz: %.0f \n", tx, ty, tz, cx, cy, cz);
        //printf("tx: %.0f ty: %.0f tz: %.0f          d2c: %.0f\n", tx, ty, tz, d2c);
        
        
        
        if (d2c >= 51 && d2c <= 54 ) {
            cnt += 1;
        }
    }
    scores[d_ix*100*400+d_iy*400+d_iz] = cnt;
}

__global__ void find_best_score_fine (int* scores,
                                float* xyz_limits,
                                float* device_pred_xyz) {
    int c_best = 0;
    device_pred_xyz[0] = -10000;
    device_pred_xyz[1] = -10000;
    device_pred_xyz[2] = -10000;


    int ixmax = int((xyz_limits[1] - xyz_limits[0])/2);
    if (ixmax > 100) ixmax = 100;
    int iymax = int((xyz_limits[3] - xyz_limits[2])/2);
    if (iymax > 100) iymax = 100;
    int izmax = int((xyz_limits[5] - xyz_limits[4])/2);
    if (izmax > 400) izmax = 400;
    printf("ixmax : %d;  iymax : %d;  izmax : %d\n", ixmax, iymax, izmax);

    for (int ix = 0; ix < ixmax; ix++) {
        for (int iy = 0; iy < iymax; iy++) {
            for (int iz = 0; iz < izmax; iz++) {
                //c_best = c_best > scores[ix*100*400+iy*400+iz] ? c_best : scores[ix*100*400+iy*400+iz];
                if (c_best < scores[ix*100*400+iy*400+iz]) {
                    c_best = scores[ix*100*400+iy*400+iz];
                    device_pred_xyz[0] = xyz_limits[0] + 5*ix;
                    device_pred_xyz[1] = xyz_limits[2] + 5*iy;
                    device_pred_xyz[2] = xyz_limits[4] + 5*iz;
                    //printf("Score: %d    x: %.0f    y: %.0f      z:%.0f \n", c_best, device_pred_xyz[0], device_pred_xyz[1], device_pred_xyz[2]);
                }
                
            }
        }
    }
}



void find_loc_fine(float* pts, int ptnum, int* scores, float* xyz_limits, float* device_pred_xyz) {

    
    //dim3 grid(10, 100, 1);
    //dim3 block(10, 1, 400);
    
    dim3 grid(100, 10, 8);
    dim3 block(1, 10, 50);
    para_find_loc_fine<<<grid, block>>>(pts, ptnum, scores, xyz_limits);


    find_best_score<<<1, 1>>>(scores, xyz_limits, device_pred_xyz);





    

    hipDeviceSynchronize();
}